#include "hip/hip_runtime.h"
/* Copyright (c) 2010 by CodeSourcery.  All rights reserved. */

/// Description
///   CUDA kernel for vector add-multiply.

#include <hip/hip_complex.h>
#include "util.hpp"
#include <vsip/support.hpp>
#include <complex>

#include "cmplx.cuh"

using namespace dev;

__global__ void 
k_am_ss(float const* in1, float const* in2, float const* in3, float* out, size_t length)
{
  int const tx = threadIdx.x;
  int const bx = blockIdx.x;

  int const idx = __mul24(blockDim.x, bx) + tx;
  if (idx < length)
  {
    out[idx] = (in1[idx] + in2[idx]) * in3[idx];
  }
}

__global__ void 
k_am_cc(hipComplex const* in1, hipComplex const* in2, hipComplex const* in3, hipComplex* out, size_t length)
{
  int const tx = threadIdx.x;
  int const bx = blockIdx.x;

  int const idx = __mul24(blockDim.x, bx) + tx;
  if (idx < length)
  {
    hipComplex tmp;
    tmp.x = in1[idx].x + in2[idx].x;
    tmp.y = in1[idx].y + in2[idx].y;
    cmul(out[idx], tmp, in3[idx]);
  }
}



namespace vsip
{
namespace impl
{
namespace cuda
{

void
am(
  float const*     in1,
  float const*     in2,
  float const*     in3,
  float*           out,
  length_type      length)
{
  dim3 grid, threads;
  distribute_vector(length, grid, threads);

  k_am_ss<<<grid, threads>>>(in1, in2, in3, out, length);
}

void
am(
  std::complex<float> const* in1,
  std::complex<float> const* in2,
  std::complex<float> const* in3,
  std::complex<float>*       out,
  length_type                length)
{
  dim3 grid, threads;
  distribute_vector(length, grid, threads);

  k_am_cc<<<grid, threads>>>(reinterpret_cast<hipComplex const *>(in1),
			     reinterpret_cast<hipComplex const *>(in2),
			     reinterpret_cast<hipComplex const *>(in3),
			     reinterpret_cast<hipComplex*>(out),
			     length);
}

} // namespace vsip::impl::cuda
} // namespace vsip::impl
} // namespace vsip
