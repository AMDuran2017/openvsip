#include "hip/hip_runtime.h"
/* Copyright (c) 2010 by CodeSourcery.  All rights reserved. */

/// Description
///   CUDA kernel for complex conjugate.

#include <hip/hip_complex.h>
#include "util.hpp"
#include <vsip/support.hpp>
#include <complex>

#include "cmplx.cuh"

namespace dev
{
// 1-D unit stride
__global__ void
conj(hipComplex const *in, hipComplex *out, size_t length)
{
  int const tx = threadIdx.x;
  int const bx = blockIdx.x;

  int const tid = __mul24(blockDim.x, bx) + tx;
  if (tid < length)
    cconj(out[tid], in[tid]);
}

// 1-D general stride
__global__ void
conj(hipComplex const *in, ptrdiff_t in_stride, hipComplex *out, ptrdiff_t out_stride, size_t length)
{
  int const tx = threadIdx.x;
  int const bx = blockIdx.x;

  int const tid = __mul24(blockDim.x, bx) + tx;
  int const in_idx = __mul24(tid, in_stride);
  int const out_idx = __mul24(tid, out_stride);
  if (tid < length)
    cconj(out[out_idx], in[in_idx]);
}

// 2-D general stride
__global__ void
conj(
  hipComplex const* in,
  ptrdiff_t        row_in_stride,
  ptrdiff_t        col_in_stride,
  hipComplex*       out,
  ptrdiff_t        row_out_stride,
  ptrdiff_t        col_out_stride,
  size_t           num_rows,
  size_t           num_cols)
{
  int const tx = threadIdx.x;
  int const ty = threadIdx.y;
  int const bx = blockIdx.x;
  int const by = blockIdx.y;

  int const tidx = __mul24(blockDim.x, bx) + tx;
  int const tidy = __mul24(blockDim.y, by) + ty;
  int const in_idx = __mul24(tidy, row_in_stride) + __mul24(tidx, col_in_stride);
  int const out_idx = __mul24(tidy, row_out_stride) + __mul24(tidx, col_out_stride);
  if (tidy < num_rows && tidx < num_cols)
    cconj(out[out_idx], in[in_idx]);
}
}// namespace dev

namespace vsip
{
namespace impl
{
namespace cuda
{
void
conj(
  std::complex<float> const*    in,
  std::complex<float>*          out,
  length_type                   length)
{
  dim3 grid, threads;
  distribute_vector(length, grid, threads);

  dev::conj<<<grid, threads>>>(reinterpret_cast<hipComplex const*>(in),
                               reinterpret_cast<hipComplex*>(out), length);
}

void
conj(
  std::complex<float> const*    in,
  stride_type                   in_stride,
  std::complex<float>*          out,
  stride_type                   out_stride,
  length_type                   length)
{
  dim3 grid, threads;
  distribute_vector(length, grid, threads);

  dev::conj<<<grid, threads>>>(reinterpret_cast<hipComplex const*>(in), in_stride,
                               reinterpret_cast<hipComplex*>(out), out_stride,
                               length);
}

void
conj(
  std::complex<float> const*    in,
  stride_type                   row_in_stride,
  stride_type                   col_in_stride,
  std::complex<float>*          out,
  stride_type                   row_out_stride,
  stride_type                   col_out_stride,
  length_type                   num_rows,
  length_type                   num_cols)
{
  dim3 grid, threads;
  distribute_matrix(num_rows, num_cols, grid, threads);

  dev::conj<<<grid, threads>>>(reinterpret_cast<hipComplex const*>(in),
                               row_in_stride, col_in_stride,
                               reinterpret_cast<hipComplex*>(out),
                               row_out_stride, col_out_stride,
                               num_rows, num_cols);
}
}// namespace vsip::impl::cuda
}// namespace vsip::impl
}// namespace vsip
