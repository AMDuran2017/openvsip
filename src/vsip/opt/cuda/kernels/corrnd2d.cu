#include "hip/hip_runtime.h"
/* Copyright (c) 2010 by CodeSourcery.  All rights reserved. */

/// Description
///   CUDA kernels for optimized 2D convolution.
#include <hip/hip_complex.h>
#include <complex>
#include <cmath>
#include "cmplx.cuh"
#include "util.hpp"

using namespace dev;

//  The CUDA computation of the 2-D correlation uses multiple stages in which
// the results for different "tiles" are computed.  The entire computation is
// separated into tiles representing:
//  a: the center region for which there is no zero padding required
//  b: the corner regions in which zero padding is required in both dimensions
//  c: the edges along the first dimension in which zero padding is required
//     along the row but not along the columns
//  d: the edges along the second dimension in which zero padding is required
//     along the column but not along the row.

//  Unbiasing is done within each tile region.

// Type for the tile region
enum tile_region_type
{
  center,
  corners,
  left_right_edge,
  top_bottom_edge
};

// Type for the support type including the even-ness of the kernel size
enum supp_region_type
{
  full_supprt,
  same_supprt_nrows_even_ncols_even,
  same_supprt_nrows_odd_ncols_even,
  same_supprt_nrows_even_ncols_odd,
  same_supprt_nrows_odd_ncols_odd,
  min_supprt
};

// Shared memory amount to be determined at runtime
extern __shared__ char shared_array_s[];

// Device function to perform the copy of data from global memory to shared
//  memory depending on "tile".
template<typename T>
__device__ inline void
d_load_global_to_shared(
  T const* input,
  T const* kernel,
  T*       input_sh,
  T*       kernel_sh,
  int      kernel_nrows,
  int      kernel_ncols,
  int      input_nrows,
  int      input_ncols,
  tile_region_type tile)
{
  int tr = threadIdx.x;
  int tc = threadIdx.y;
  int tidr = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
  int tidc = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
  int width = __mul24(2, blockDim.y);

  if (tile == center)
  {
    if (tr < kernel_nrows && tc < kernel_ncols)
      kernel_sh[__mul24(tr, blockDim.y) + tc] =
         kernel[__mul24(tr, kernel_ncols) + tc];

    if (tidr < input_nrows && tidc < input_ncols)
      input_sh[__mul24(tr, width) + tc] =
         input[__mul24(tidr, input_ncols) + tidc];

    if (tidr + blockDim.x < input_nrows && tidc < input_ncols)
      input_sh[__mul24(tr + blockDim.x, width) + tc] =
         input[__mul24(tidr + blockDim.x, input_ncols) + tidc];

    if (tidr < input_nrows && tidc + blockDim.y < input_ncols)
      input_sh[__mul24(tr, width) + tc + blockDim.y] =
         input[__mul24(tidr, input_ncols) + tidc + blockDim.y];

    if (tidr + blockDim.x < input_nrows && tidc + blockDim.y < input_ncols)
      input_sh[__mul24(tr + blockDim.x, width) + tc + blockDim.y] =
         input[__mul24(tidr + blockDim.x, input_ncols) + tidc + blockDim.y];
  }
  else if (tile == corners)
  {
    kernel_sh[__mul24(tr, blockDim.y) + tc] =
       kernel[__mul24(tr, kernel_ncols) + tc];

    input_sh[__mul24(tr, width) + tc] = input[__mul24(tr, input_ncols) + tc];

    input_sh[__mul24(tr + kernel_nrows, width) + tc] =
       input[__mul24(tr + input_nrows - kernel_nrows, input_ncols) + tc];

    input_sh[__mul24(tr, width) + tc + kernel_ncols] =
       input[__mul24(tr, input_ncols) + tc + input_ncols - kernel_ncols];

    input_sh[__mul24(tr + kernel_nrows, width) + tc + kernel_ncols] =
       input[__mul24(tr + input_nrows - kernel_nrows, input_ncols) +
             tc + input_ncols - kernel_ncols];
  }
  else if (tile == left_right_edge)
  {
    if (tr < kernel_nrows)
      kernel_sh[__mul24(tr, blockDim.y) + tc] =
         kernel[__mul24(tr, kernel_ncols) + tc];

    if (tidr < input_nrows)
    {
      input_sh[__mul24(tr, width) + tc] =
         input[__mul24(tidr, input_ncols) + tc];

      input_sh[__mul24(tr, width) + tc + kernel_ncols] =
         input[__mul24(tidr, input_ncols) + tc + input_ncols - kernel_ncols];
    }

    if (tidr + blockDim.x < input_nrows)
    {
      input_sh[__mul24(tr + blockDim.x, width) + tc] =
         input[__mul24(tidr + blockDim.x, input_ncols) + tc];

      input_sh[__mul24(tr + blockDim.x, width) + tc + kernel_ncols] =
         input[__mul24(tidr + blockDim.x, input_ncols) +
               tc + input_ncols - kernel_ncols];
    }
  }
  else if (tile == top_bottom_edge)
  {
    if (tc < kernel_ncols)
      kernel_sh[__mul24(tr, blockDim.y) + tc] =
         kernel[__mul24(tr, kernel_ncols) + tc];

    if (tidc < input_ncols)
    {
      input_sh[__mul24(tr, width) + tc] =
         input[__mul24(tr, input_ncols) + tidc];

      input_sh[__mul24(tr + kernel_nrows, width) + tc] =
         input[__mul24(tr + input_nrows - kernel_nrows, input_ncols) + tidc];
    }

    if (tidc + blockDim.y < input_ncols)
    {
      input_sh[__mul24(tr, width) + tc + blockDim.y] =
         input[__mul24(tr, input_ncols) + tidc + blockDim.y];

      input_sh[__mul24(tr + kernel_nrows, width) + tc + blockDim.y] =
      input[__mul24(tr + input_nrows - kernel_nrows, input_ncols) +
            tidc + blockDim.y];
    }
  }
}

template void d_load_global_to_shared<float>(
                  float const*, float const*, float*, float*, int, int, int,
                  int, tile_region_type);

template void d_load_global_to_shared<hipComplex>(
                  hipComplex const*, hipComplex const*, hipComplex*, hipComplex*,
                  int, int, int, int, tile_region_type);

//  Device function to store a value or set of values ("sum_x") to the
//   appropriate global memory region depending on "tile" and "supp"
template<typename T>
__device__ inline void
d_store_var_to_global(
  T*       out,
  T        sum,
  T        sum_begin,
  T        sum_end,
  T        sum_ur,
  T        sum_ul,
  T        sum_ll,
  T        sum_lr,
  int      kernel_nrows,
  int      kernel_ncols,
  int      input_nrows,
  int      input_ncols,
  int      output_nrows,
  int      output_ncols,
  int      row_shift,
  int      col_shift,
  tile_region_type tile,
  supp_region_type supp)
{
  int tr = threadIdx.x;
  int tc = threadIdx.y;
  int tidr = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
  int tidc = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;

  if (tile == center)
  {
    if (supp == full_supprt)
    {
      if (tidc < input_ncols - kernel_ncols + 1 &&
          tidr < input_nrows - kernel_nrows + 1)
        *(out + __mul24(tidr + kernel_nrows - 1, output_ncols) +
                tidc + kernel_ncols - 1) = sum;
    }
    else if (supp == same_supprt_nrows_even_ncols_even)
    {
      if (tidc < input_ncols - kernel_ncols + 1 &&
          tidr < input_nrows - kernel_nrows + 1)
        *(out + __mul24(tidr - row_shift, output_ncols) +
                tidc - col_shift) = sum;
    }
    else if (supp == same_supprt_nrows_even_ncols_odd)
    {
      if (tidc < input_ncols - kernel_ncols + 1 &&
          tidr < input_nrows - kernel_nrows + 1)
        *(out + __mul24(tidr - row_shift, output_ncols) +
                tidc - col_shift) = sum;
    }
    else if (supp == same_supprt_nrows_odd_ncols_even)
    {
      if (tidc < input_ncols - kernel_ncols + 1 &&
          tidr < input_nrows - kernel_nrows + 1)
        *(out + __mul24(tidr - row_shift, output_ncols) +
                tidc - col_shift) = sum;
    }
    else if (supp == same_supprt_nrows_odd_ncols_odd)
    {
      if (tidc < input_ncols - kernel_ncols + 1 &&
          tidr < input_nrows - kernel_nrows + 1)
        *(out + __mul24(tidr - row_shift, output_ncols) +
                tidc - col_shift) = sum;
    }
    else if (supp == min_supprt)
    {
      if (tidc < output_ncols && tidr < output_nrows)
        *(out + __mul24(tidr, output_ncols) + tidc) = sum;
    }
  }
  else if (tile == corners)
  {
    if (supp == full_supprt)
    {
      if (tr < kernel_nrows - 1 && tc < kernel_ncols - 1)
      {
        out[__mul24(tr, output_ncols) + tc] = sum_ul;
        out[__mul24(output_nrows - tr - 1, output_ncols) + tc] = sum_ll;
        out[__mul24(tr, output_ncols) + output_ncols - tc - 1] = sum_ur;
        out[__mul24(output_nrows - tr - 1, output_ncols) +
            output_ncols - tc - 1] = sum_lr;
      }
    }
    else if (supp == same_supprt_nrows_even_ncols_even)
    {
      if (tr == 0 && tc == 0)
        out[__mul24(tr, output_ncols) +
            tc] = sum_ul;
      else if (tr == 0 && tc < kernel_ncols + col_shift)
      {
        out[__mul24(tr, output_ncols) + tc] = sum_ul;
        out[__mul24(tr, output_ncols) +
            output_ncols - tc] = sum_ur;
      }
      else if (tr < kernel_nrows + row_shift && tc == 0)
      {
        out[__mul24(tr, output_ncols) + tc] = sum_ul;
        out[__mul24(output_nrows - tr, output_ncols) +
            tc] = sum_ll;
      }
      else if (tr < kernel_nrows + row_shift && tr > 0 &&
               tc < kernel_ncols + col_shift && tc > 0)
      {
        out[__mul24(tr, output_ncols) + tc] = sum_ul;
        out[__mul24(output_nrows - tr, output_ncols) + tc] = sum_ll;
        out[__mul24(tr, output_ncols) + output_ncols - tc] = sum_ur;
        out[__mul24(output_nrows - tr, output_ncols) +
            output_ncols - tc] = sum_lr;
      }
    }
    else if (supp == same_supprt_nrows_even_ncols_odd)
    {
      if (tc < kernel_ncols + col_shift - 1)
      {
        if (tr == 0)
        {
          out[__mul24(tr, output_ncols) + tc] = sum_ul;
          out[__mul24(tr, output_ncols) +
              output_ncols - tc - 1] = sum_ur;
        }
        else if (tr < kernel_nrows + row_shift)
        {
          out[__mul24(tr, output_ncols) + tc] = sum_ul;
          out[__mul24(output_nrows - tr, output_ncols) + tc] = sum_ll;
          out[__mul24(tr, output_ncols) + output_ncols - tc - 1] = sum_ur;
          out[__mul24(output_nrows - tr, output_ncols) +
              output_ncols - tc - 1] = sum_lr;
        }
      }
    }
    else if (supp == same_supprt_nrows_odd_ncols_even)
    {
      if (tr < kernel_nrows + row_shift - 1)
      {
        if (tc == 0)
        {
          out[__mul24(tr, output_ncols) + tc] = sum_ul;
          out[__mul24(output_nrows - tr - 1, output_ncols) +
              tc] = sum_ll;
        }
        else if (tc < kernel_ncols + col_shift)
        {
          out[__mul24(tr, output_ncols) + tc] = sum_ul;
          out[__mul24(output_nrows - tr - 1, output_ncols) + tc] = sum_ll;
          out[__mul24(tr, output_ncols) + output_ncols - tc] = sum_ur;
          out[__mul24(output_nrows - tr - 1, output_ncols) +
              output_ncols - tc] = sum_lr;
        }
      }
    }
    else if (supp == same_supprt_nrows_odd_ncols_odd)
    {
      if (tr < kernel_nrows + row_shift - 1 &&
          tc < kernel_ncols + col_shift - 1)
      {
        out[__mul24(tr, output_ncols) + tc] = sum_ul;
        out[__mul24(output_nrows - tr - 1, output_ncols) + tc] = sum_ll;
        out[__mul24(tr, output_ncols) + output_ncols - tc - 1] = sum_ur;
        out[__mul24(output_nrows - tr - 1, output_ncols) +
            output_ncols - tc - 1] = sum_lr;
      }
    }
  }
  else if (tile == left_right_edge)
  {
    if (supp == full_supprt)
    {
      if (tidr < input_nrows - kernel_nrows + 1 && tc < kernel_ncols - 1)
      {
        out[__mul24(tidr + kernel_nrows - 1, output_ncols) + tc] = sum_begin;
        out[__mul24(tidr + kernel_nrows - 1, output_ncols) +
            output_ncols - tc - 1] = sum_end;
      }
    }
    else if (supp == same_supprt_nrows_even_ncols_even)
    {
      if (tidr < input_nrows - kernel_nrows + 1)
      {
        if (tc == 0)
          out[__mul24(tidr - row_shift, output_ncols) +
              tc] = sum_begin;
        else if (tc < kernel_ncols + col_shift && tc > 0)
        {
          out[__mul24(tidr - row_shift, output_ncols) + tc] = sum_begin;
          out[__mul24(tidr - row_shift, output_ncols) +
              output_ncols - tc] = sum_end;
        }
      }
    }
    else if (supp == same_supprt_nrows_even_ncols_odd)
    {
      if (tidr < input_nrows - kernel_nrows + 1 &&
            tc < kernel_ncols + col_shift - 1)
      {
        out[__mul24(tidr - row_shift, output_ncols) + tc] = sum_begin;
        out[__mul24(tidr - row_shift, output_ncols) +
                    output_ncols - tc - 1] = sum_end;
      }
    }
    else if (supp == same_supprt_nrows_odd_ncols_even)
    {
      if (tidr < input_nrows - kernel_nrows + 1)
      {
        if (tc == 0)
          out[__mul24(tidr - row_shift, output_ncols) +
              tc] = sum_begin;
        else if (tc < kernel_ncols + col_shift)
        {
          out[__mul24(tidr - row_shift, output_ncols) + tc] = sum_begin;
          out[__mul24(tidr - row_shift, output_ncols) +
              output_ncols - tc] = sum_end;
        }
      }
    }
    else if (supp == same_supprt_nrows_odd_ncols_odd)
    {
      if (tidr < input_nrows - kernel_nrows + 1 &&
            tc < kernel_ncols + col_shift - 1)
      {
        out[__mul24(tidr - row_shift, output_ncols) + tc] = sum_begin;
        out[__mul24(tidr - row_shift, output_ncols) +
            output_ncols - tc - 1] = sum_end;
      }
    }
  }
  else if (tile == top_bottom_edge)
  {
    if (supp == full_supprt)
    {
      if (tr < kernel_nrows - 1 && tidc < input_ncols - kernel_ncols + 1)
      {
        out[__mul24(tr, output_ncols) + tidc + kernel_ncols - 1] = sum_begin;
        out[__mul24(output_nrows - tr - 1, output_ncols) +
            tidc + kernel_ncols - 1] = sum_end;
      }
    }
    else if (supp == same_supprt_nrows_even_ncols_even)
    {
      if (tidc < input_ncols - kernel_ncols + 1)
      {
        if (tr == 0)
          out[__mul24(tr, output_ncols) +
              tidc - col_shift] = sum_begin;
        else if (tr < kernel_nrows + row_shift && tr > 0)
        {
          out[__mul24(tr, output_ncols) + tidc - col_shift] = sum_begin;

          out[__mul24(output_nrows - tr, output_ncols) +
              tidc - col_shift] = sum_end;
        }
      }
    }
    else if (supp == same_supprt_nrows_even_ncols_odd)
    {
      if (tidc < input_ncols - kernel_ncols + 1)
      {
        if (tr == 0)
          out[__mul24(tr, output_ncols) +
              tidc - col_shift] = sum_begin;
        else if (tr < kernel_nrows + row_shift)
        {
          out[__mul24(tr, output_ncols) + tidc - col_shift] = sum_begin;
          out[__mul24(output_nrows - tr, output_ncols) +
              tidc - col_shift] = sum_end;
        }
      }
    }
    else if (supp == same_supprt_nrows_odd_ncols_even)
    {
      if (tidc < input_ncols - kernel_ncols + 1 &&
            tr < kernel_nrows - 1 + row_shift)
      {
        out[__mul24(tr, output_ncols) + tidc - col_shift] = sum_begin;
        out[__mul24(output_nrows - tr - 1, output_ncols) +
            tidc - col_shift] = sum_end;
      }
    }
    else if (supp == same_supprt_nrows_odd_ncols_odd)
    {
      if (tidc < input_ncols - kernel_ncols + 1 &&
            tr < kernel_nrows + row_shift - 1)
      {
        out[__mul24(tr, output_ncols) + tidc - col_shift] = sum_begin;
        out[__mul24(output_nrows - tr - 1, output_ncols) +
            tidc - col_shift] = sum_end;
      }
    }
  }
}

template void d_store_var_to_global<float>(float*, float, float, float, float,
                                           float,  float, float, int, int, int,
                                           int, int, int, int, int,
                                           tile_region_type, supp_region_type);

template void d_store_var_to_global<hipComplex>(hipComplex*, hipComplex, hipComplex,
                                               hipComplex,  hipComplex, hipComplex,
                                               hipComplex,  hipComplex, int, int,
                                               int, int, int, int, int, int,
                                               tile_region_type, supp_region_type);


// Device function to perform the calculation by looping over rows and columns.
//  The result(s) are stored in the "sum" variables.
template<typename T>
__device__ inline void
d_loops(
  T*       input_sh,
  T*       kernel_sh,
  T&        sum,
  T&        sum_begin,
  T&        sum_end,
  T&        sum_ur,
  T&        sum_ul,
  T&        sum_ll,
  T&        sum_lr,
  int      kernel_nrows,
  int      kernel_ncols,
  int      input_nrows,
  int      input_ncols,
  int      output_nrows,
  int      output_ncols,
  int      row_shift,
  int      col_shift,
  int      delta_tr,
  int      delta_tc,
  int      is_unbiased,
  tile_region_type tile,
  supp_region_type supp)
{
  int tr = threadIdx.x;
  int tc = threadIdx.y;
  int tidr = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
  int tidc = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
  int width = __mul24(2, blockDim.y);
  T temp;

  if (tile == center)
  {
    for (int i = 0; i < kernel_nrows; ++i)
    {
      T sum_row;

      sum_row *= 0.0F;
      for (int j = 0; j < kernel_ncols; ++j)
      {
        int row_index = tr + i;
        int col_index = tc + j;

        cconj(temp, input_sh[__mul24(row_index, width) + col_index]);

        sum_row += temp * kernel_sh[__mul24(i, blockDim.y) + j];
      }
      sum += sum_row;
    }

    if (is_unbiased)
      sum /= float(kernel_nrows * kernel_ncols);
  }
  else if (tile == corners)
  {
    for (int i = 0; i <= delta_tr; ++i)
    {
      T sum_row_ul, sum_row_ur, sum_row_lr, sum_row_ll;

      sum_row_ul *= 0.0;
      sum_row_ur *= 0.0;
      sum_row_lr *= 0.0;
      sum_row_ll *= 0.0;  

      for (int j = 0; j <= delta_tc; ++j)
      {
        cconj(temp, input_sh[__mul24(i, width) + j]);

        sum_row_ul += temp * kernel_sh[__mul24(i - row_shift - tr, blockDim.y) + j - col_shift - tc];

        cconj(temp, input_sh[__mul24(i, width) + __mul24(kernel_ncols, 2) - delta_tc + j - 1]);

        sum_row_ur += temp * kernel_sh[__mul24(i - row_shift - tr, blockDim.y) + j];

        cconj(temp, input_sh[(__mul24(kernel_nrows, 2) - delta_tr + i - 1) * width + j]);

        sum_row_ll += temp * kernel_sh[__mul24(i, blockDim.y) + j - col_shift - tc];

        cconj(temp, input_sh[(__mul24(kernel_nrows, 2) - delta_tr + i - 1) * width + __mul24(kernel_ncols, 2) - delta_tc + j - 1]);

        sum_row_lr += temp * kernel_sh[__mul24(i, blockDim.y) + j];
      }
      sum_ul += sum_row_ul;
      sum_ur += sum_row_ur;
      sum_ll += sum_row_ll;
      sum_lr += sum_row_lr;
    }
 
    if (is_unbiased)
    {
      sum_ul /= float((tr + 1) * (tc + 1));
      sum_ur /= float((tr + 1) * (tc + 1));
      sum_ll /= float((tr + 1) * (tc + 1));
      sum_lr /= float((tr + 1) * (tc + 1));
    }
  }
  else if (tile == left_right_edge)
  {
    for (int i = 0; i < kernel_nrows; ++i)
    {
      T sum_row_begin, sum_row_end;

      sum_row_begin *= 0.0;
      sum_row_end *= 0.0;

      for (int j = 0; j <= delta_tc; ++j)
      {
        cconj(temp, input_sh[__mul24(tr + i, width) + j]);

        sum_row_begin += temp * kernel_sh[__mul24(i, blockDim.y) + j - col_shift - tc];

        cconj(temp, input_sh[__mul24(tr + i, width) + __mul24(kernel_ncols, 2) - delta_tc + j - 1]);

        sum_row_end += temp * kernel_sh[__mul24(i, blockDim.y) + j];
      }
      sum_begin += sum_row_begin;
      sum_end += sum_row_end;
    }

    if (is_unbiased)
    {
      sum_begin /= float(kernel_nrows * (tc + 1));
      sum_end /= float(kernel_nrows * (tc + 1));
    }
  }
  else if (tile == top_bottom_edge)
  {
    for (int j = 0; j < kernel_ncols; ++j)
    {
      T sum_col_begin, sum_col_end;

      sum_col_begin *= 0.0;
      sum_col_end *= 0.0;

      for (int i = 0; i <= delta_tr; ++i)
      {
        cconj(temp, input_sh[__mul24(i, width) + tc + j]);

        sum_col_begin += temp * kernel_sh[__mul24(i - row_shift - tr, blockDim.y) + j];

        cconj(temp, input_sh[(__mul24(kernel_nrows, 2) - delta_tr + i - 1) * width + tc + j]);

        sum_col_end += temp * kernel_sh[__mul24(i, blockDim.y) + j];
      }
      sum_begin += sum_col_begin;
      sum_end += sum_col_end;
    }

    if (is_unbiased)
    {
      sum_begin /= float(kernel_ncols * (tr + 1));
      sum_end /= float(kernel_ncols * (tr + 1));
    }
  }
}

template void d_loops<float>(float*, float*, float&, float&, float&,
                             float&,  float&, float&, float&, int, int, int,
                             int, int, int, int, int, int, int, int,
                             tile_region_type, supp_region_type);

template void d_loops<hipComplex>(hipComplex*, hipComplex*, hipComplex&,
                                 hipComplex&,  hipComplex&, hipComplex&,
                                 hipComplex&,  hipComplex&, hipComplex&, int, int,
                                 int, int, int, int, int, int, int, int, int,
                                 tile_region_type, supp_region_type);

// Global kernel function to perform 2-D convolution
template<typename T>
__global__ void
k_corr2dnd(
  T const* input,
  T const* kernel,
  T*       out,
  int          input_nrows,
  int          input_ncols,
  int          kernel_nrows,
  int          kernel_ncols,
  int          output_nrows,
  int          output_ncols,
  int          row_shift,
  int          col_shift,
  int          del_tr,
  int          del_tc,
  int                 bias,
  tile_region_type tile,
  supp_region_type supp)
{
  int tr = threadIdx.x;
  int tc = threadIdx.y;
  int br = blockIdx.x;
  int bc = blockIdx.y;
  int tidr = __mul24(blockDim.x, br) + tr;
  int tidc = __mul24(blockDim.y, bc) + tc;
  int delta_tr = tr + del_tr;
  int delta_tc = tc + del_tc;

  T sum, sum_begin, sum_end, sum_ur, sum_ul, sum_ll, sum_lr;

  sum *= 0.0;
  sum_begin *= 0.0;
  sum_end *= 0.0;
  sum_ur *= 0.0;
  sum_ul *= 0.0;
  sum_ll *= 0.0;
  sum_lr *= 0.0;

  T *kernel_sh = (T*)shared_array_s;
  T *input_sh = (T*)&shared_array_s[blockDim.x * blockDim.y *
                                    sizeof(T) / sizeof(char)];

  d_load_global_to_shared<T>(input, kernel, input_sh, kernel_sh, kernel_nrows,
                             kernel_ncols, input_nrows, input_ncols, tile);

  __syncthreads();

  d_loops<T>(input_sh, kernel_sh, sum, sum_begin, sum_end, sum_ur, sum_ul,
             sum_ll, sum_lr, kernel_nrows, kernel_ncols, input_nrows,
             input_ncols, output_nrows, output_ncols, row_shift, col_shift,
             delta_tr, delta_tc, bias, tile, supp);

  d_store_var_to_global<T>(out, sum, sum_begin, sum_end, sum_ur, sum_ul,
                           sum_ll, sum_lr, kernel_nrows, kernel_ncols,
                           input_nrows, input_ncols, output_nrows,
                           output_ncols, row_shift, col_shift, tile, supp);
}



namespace vsip
{
namespace impl
{
namespace cuda
{

// Kernels are launched with 16000 bytes of shared memory per block in order
//  to keep enough shared memory to hold the data but to also leave enough
//  room for function arguments, static memory, and execution configuration.
// The number of threads to launch is based on the use of a A x A/2 size
//  array in shared memory for the kernel thus requiring a maximum 2A x A
//  size array for the input.  Thus the required amount of shared memory is
//  (A^2)/2 + 2(A^2) elements = 2.5A^2 elements.
void
corr_2d_no_decimation_min(
  float const*        in,
  float const*        kr,
  float*              out,
  size_t              in_nr,
  size_t              in_nc,
  size_t              kr_nr,
  size_t              kr_nc,
  size_t              ou_nr,
  size_t              ou_nc,
  size_t              rshift,
  size_t              cshift,
  int                 bias)
{
  dim3 grid, threads;

  int dtr = 0;
  int dtc = 0;

  size_t const shared_memory_launch_size = 16000;

  distribute_matrix(ou_nc, ou_nr, grid, threads);

  k_corr2dnd<<<grid, threads, shared_memory_launch_size>>>(
               in, kr, out, int(in_nr), int(in_nc), int(kr_nr), int(kr_nc),
               int(ou_nr), int(ou_nc), int(rshift), int(cshift), dtr,
               dtc, bias, center, min_supprt);

}

void
corr_2d_no_decimation_min(
  std::complex<float> const*        in,
  std::complex<float> const*        kr,
  std::complex<float>*              out,
  size_t              in_nr,
  size_t              in_nc,
  size_t              kr_nr,
  size_t              kr_nc,
  size_t              ou_nr,
  size_t              ou_nc,
  size_t              rshift,
  size_t              cshift,
  int                 bias)
{
  dim3 grid, threads;

  int dtr = 0;
  int dtc = 0;

  size_t const shared_memory_launch_size = 16000;

  threads.x = min(int(Dev_props::max_threads_per_block_x()),
                  int(sqrt(float(Dev_props::shared_memory_size() - 72) /
                                (8.0 * 2.5))));

  threads.y = threads.x / 2;

  grid.x = (in_nr - kr_nr + threads.x) / threads.x;
  grid.y = (in_nc - kr_nc + threads.y) / threads.y;

  k_corr2dnd<<<grid, threads, shared_memory_launch_size>>>(
              reinterpret_cast<hipComplex const*>(in),
              reinterpret_cast<hipComplex const*>(kr),
              reinterpret_cast<hipComplex*>(out), int(in_nr), int(in_nc),
              int(kr_nr), int(kr_nc), int(ou_nr), int(ou_nc), int(rshift),
              int(cshift), dtr, dtc, bias, center, min_supprt);

}

void
corr_2d_no_decimation_full(
  float const*        in,
  float const*        kr,
  float*              out,
  size_t              in_nr,
  size_t              in_nc,
  size_t              kr_nr,
  size_t              kr_nc,
  size_t              ou_nr,
  size_t              ou_nc,
  size_t              rshift,
  size_t              cshift,
  int                 bias)
{
  dim3 grid, threads, grid_corner, threads_corner, grid_lr_edge,
       threads_lr_edge, grid_tb_edge, threads_tb_edge;

  int dtr = 0;
  int dtc = 0;

  size_t const shared_memory_launch_size = 16000;

  distribute_matrix(in_nc - kr_nc + 1, in_nr - kr_nr + 1, grid, threads);
  grid_corner.x = 1;
  grid_corner.y = 1;
  threads_corner.x = kr_nr;
  threads_corner.y = kr_nc;

  grid_lr_edge.x = (in_nr - kr_nr + int(Dev_props::max_threads_per_block_x())) /
                                    int(Dev_props::max_threads_per_block_x());
  grid_lr_edge.y = 1;

  grid_tb_edge.x = 1;
  grid_tb_edge.y = (in_nc - kr_nc + int(Dev_props::max_threads_per_block_y())) /
                                    int(Dev_props::max_threads_per_block_y());

  threads_lr_edge.x = int(Dev_props::max_threads_per_block_x());
  threads_lr_edge.y = kr_nc;

  threads_tb_edge.x = kr_nr;
  threads_tb_edge.y = int(Dev_props::max_threads_per_block_y());

  k_corr2dnd<<<grid, threads, shared_memory_launch_size>>>(
               in, kr, out, int(in_nr), int(in_nc), int(kr_nr), int(kr_nc),
               int(ou_nr), int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               center, full_supprt);

  k_corr2dnd<<<grid_corner, threads_corner, shared_memory_launch_size>>>(
               in, kr, out, int(in_nr), int(in_nc), int(kr_nr), int(kr_nc),
               int(ou_nr), int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               corners, full_supprt);

  k_corr2dnd<<<grid_lr_edge, threads_lr_edge, shared_memory_launch_size>>>(
               in, kr, out, int(in_nr), int(in_nc), int(kr_nr), int(kr_nc),
               int(ou_nr), int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               left_right_edge, full_supprt);

  k_corr2dnd<<<grid_tb_edge, threads_tb_edge, shared_memory_launch_size>>>(
               in, kr, out, int(in_nr), int(in_nc), int(kr_nr), int(kr_nc),
               int(ou_nr), int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               top_bottom_edge, full_supprt);

}

void
corr_2d_no_decimation_full(
  std::complex<float> const*        in,
  std::complex<float> const*        kr,
  std::complex<float>*              out,
  size_t              in_nr,
  size_t              in_nc,
  size_t              kr_nr,
  size_t              kr_nc,
  size_t              ou_nr,
  size_t              ou_nc,
  size_t              rshift,
  size_t              cshift,
  int                 bias)
{
  dim3 grid, threads, grid_corner, threads_corner, grid_lr_edge,
       threads_lr_edge, grid_tb_edge, threads_tb_edge;

  int dtr = 0;
  int dtc = 0;

  size_t const shared_memory_launch_size = 16000;

  threads.x = min(int(Dev_props::max_threads_per_block_x()),
                  int(sqrt(float(Dev_props::shared_memory_size() - 72) /
                                (8.0 * 2.5))));
  threads.y = threads.x / 2;


  grid.x = (in_nr - kr_nr + threads.x) / threads.x;
  grid.y = (in_nc - kr_nc + threads.y) / threads.y;



  grid_corner.x = 1;
  grid_corner.y = 1;
  threads_corner.x = kr_nr;
  threads_corner.y = kr_nc;

  grid_lr_edge.x = grid.x;
  grid_lr_edge.y = 1;

  grid_tb_edge.x = 1;
  grid_tb_edge.y = grid.y;

  threads_lr_edge.x = threads.x;
  threads_lr_edge.y = kr_nc;

  threads_tb_edge.x = kr_nr;
  threads_tb_edge.y = threads.y;

  k_corr2dnd<<<grid, threads, shared_memory_launch_size>>>(
               reinterpret_cast<hipComplex const*>(in),
               reinterpret_cast<hipComplex const*>(kr),
               reinterpret_cast<hipComplex*>(out), int(in_nr),
               int(in_nc), int(kr_nr), int(kr_nc), int(ou_nr),
               int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               center, full_supprt);

  k_corr2dnd<<<grid_corner, threads_corner, shared_memory_launch_size>>>(
               reinterpret_cast<hipComplex const*>(in),
               reinterpret_cast<hipComplex const*>(kr),
               reinterpret_cast<hipComplex*>(out), int(in_nr),
               int(in_nc), int(kr_nr), int(kr_nc), int(ou_nr),
               int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               corners, full_supprt);

  k_corr2dnd<<<grid_lr_edge, threads_lr_edge, shared_memory_launch_size>>>(
               reinterpret_cast<hipComplex const*>(in),
               reinterpret_cast<hipComplex const*>(kr),
               reinterpret_cast<hipComplex*>(out), int(in_nr),
               int(in_nc), int(kr_nr), int(kr_nc), int(ou_nr),
               int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               left_right_edge, full_supprt);

  k_corr2dnd<<<grid_tb_edge, threads_tb_edge, shared_memory_launch_size>>>(
               reinterpret_cast<hipComplex const*>(in),
               reinterpret_cast<hipComplex const*>(kr),
               reinterpret_cast<hipComplex*>(out), int(in_nr),
               int(in_nc), int(kr_nr), int(kr_nc), int(ou_nr),
               int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               top_bottom_edge, full_supprt);

}

void
corr_2d_no_decimation_same_nrow_even_ncol_even(
  float const*        in,
  float const*        kr,
  float*              out,
  size_t              in_nr,
  size_t              in_nc,
  size_t              kr_nr,
  size_t              kr_nc,
  size_t              ou_nr,
  size_t              ou_nc,
  size_t              rshift,
  size_t              cshift,
  int                 bias)
{
  dim3 grid, threads, grid_corner, threads_corner, grid_lr_edge,
       threads_lr_edge, grid_tb_edge, threads_tb_edge;

  int dtr = -rshift - 1;
  int dtc = -cshift - 1;

  size_t const shared_memory_launch_size = 16000;

  distribute_matrix(in_nc - kr_nc + 1, in_nr - kr_nr + 1, grid, threads);
  grid_corner.x = 1;
  grid_corner.y = 1;
  threads_corner.x = kr_nr;
  threads_corner.y = kr_nc;

  grid_lr_edge.x = (in_nr - kr_nr + int(Dev_props::max_threads_per_block_x())) /
                                    int(Dev_props::max_threads_per_block_x());
  grid_lr_edge.y = 1;

  grid_tb_edge.x = 1;
  grid_tb_edge.y = (in_nc - kr_nc + int(Dev_props::max_threads_per_block_y())) /
                                    int(Dev_props::max_threads_per_block_y());

  threads_lr_edge.x = int(Dev_props::max_threads_per_block_x());
  threads_lr_edge.y = kr_nc;

  threads_tb_edge.x = kr_nr;
  threads_tb_edge.y = int(Dev_props::max_threads_per_block_y());

  k_corr2dnd<<<grid, threads, shared_memory_launch_size>>>(
               in, kr, out, int(in_nr), int(in_nc), int(kr_nr), int(kr_nc),
               int(ou_nr), int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               center, same_supprt_nrows_even_ncols_even);

  k_corr2dnd<<<grid_corner, threads_corner, shared_memory_launch_size>>>(
               in, kr, out, int(in_nr), int(in_nc), int(kr_nr), int(kr_nc),
               int(ou_nr), int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               corners, same_supprt_nrows_even_ncols_even);

  k_corr2dnd<<<grid_lr_edge, threads_lr_edge, shared_memory_launch_size>>>(
               in, kr, out, int(in_nr), int(in_nc), int(kr_nr), int(kr_nc),
               int(ou_nr), int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               left_right_edge, same_supprt_nrows_even_ncols_even);

  k_corr2dnd<<<grid_tb_edge, threads_tb_edge, shared_memory_launch_size>>>(
               in, kr, out, int(in_nr), int(in_nc),  int(kr_nr), int(kr_nc),
               int(ou_nr), int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               top_bottom_edge, same_supprt_nrows_even_ncols_even);

}

void
corr_2d_no_decimation_same_nrow_even_ncol_even(
  std::complex<float> const*        in,
  std::complex<float> const*        kr,
  std::complex<float>*              out,
  size_t              in_nr,
  size_t              in_nc,
  size_t              kr_nr,
  size_t              kr_nc,
  size_t              ou_nr,
  size_t              ou_nc,
  size_t              rshift,
  size_t              cshift,
  int                 bias)
{
  dim3 grid, threads, grid_corner, threads_corner, grid_lr_edge,
       threads_lr_edge, grid_tb_edge, threads_tb_edge;

  int dtr = -rshift - 1;
  int dtc = -cshift - 1;

  size_t const shared_memory_launch_size = 16000;

  threads.x = min(int(Dev_props::max_threads_per_block_x()),
                  int(sqrt(float(Dev_props::shared_memory_size() - 72) /
                                (8.0 * 2.5))));
  threads.y = threads.x / 2;

  grid.x = (in_nr - kr_nr + threads.x) / threads.x;
  grid.y = (in_nc - kr_nc + threads.y) / threads.y;

  grid_corner.x = 1;
  grid_corner.y = 1;
  threads_corner.x = kr_nr;
  threads_corner.y = kr_nc;

  grid_lr_edge.x = grid.x;
  grid_lr_edge.y = 1;

  grid_tb_edge.x = 1;
  grid_tb_edge.y = grid.y;

  threads_lr_edge.x = threads.x;
  threads_lr_edge.y = kr_nc;

  threads_tb_edge.x = kr_nr;
  threads_tb_edge.y = threads.y;

  k_corr2dnd<<<grid, threads, shared_memory_launch_size>>>(
               reinterpret_cast<hipComplex const*>(in),
               reinterpret_cast<hipComplex const*>(kr),
               reinterpret_cast<hipComplex*>(out), int(in_nr),
               int(in_nc), int(kr_nr), int(kr_nc), int(ou_nr),
               int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               center, same_supprt_nrows_even_ncols_even);

  k_corr2dnd<<<grid_corner, threads_corner, shared_memory_launch_size>>>(
               reinterpret_cast<hipComplex const*>(in),
               reinterpret_cast<hipComplex const*>(kr),
               reinterpret_cast<hipComplex*>(out), int(in_nr),
               int(in_nc), int(kr_nr), int(kr_nc), int(ou_nr),
               int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               corners, same_supprt_nrows_even_ncols_even);

  k_corr2dnd<<<grid_lr_edge, threads_lr_edge, shared_memory_launch_size>>>(
               reinterpret_cast<hipComplex const*>(in),
               reinterpret_cast<hipComplex const*>(kr),
               reinterpret_cast<hipComplex*>(out), int(in_nr),
               int(in_nc), int(kr_nr), int(kr_nc), int(ou_nr),
               int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               left_right_edge, same_supprt_nrows_even_ncols_even);

  k_corr2dnd<<<grid_tb_edge, threads_tb_edge, shared_memory_launch_size>>>(
               reinterpret_cast<hipComplex const*>(in),
               reinterpret_cast<hipComplex const*>(kr),
               reinterpret_cast<hipComplex*>(out), int(in_nr),
               int(in_nc), int(kr_nr), int(kr_nc), int(ou_nr),
               int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               top_bottom_edge, same_supprt_nrows_even_ncols_even);

}

void
corr_2d_no_decimation_same_nrow_even_ncol_odd(
  float const*        in,
  float const*        kr,
  float*              out,
  size_t              in_nr,
  size_t              in_nc,
  size_t              kr_nr,
  size_t              kr_nc,
  size_t              ou_nr,
  size_t              ou_nc,
  size_t              rshift,
  size_t              cshift,
  int                 bias)
{
  dim3 grid, threads, grid_corner, threads_corner, grid_lr_edge,
       threads_lr_edge, grid_tb_edge, threads_tb_edge;

  int dtr = -rshift - 1;
  int dtc = -cshift;

  size_t const shared_memory_launch_size = 16000;

  distribute_matrix(in_nc - kr_nc + 1, in_nr - kr_nr + 1, grid, threads);
  grid_corner.x = 1;
  grid_corner.y = 1;
  threads_corner.x = kr_nr;
  threads_corner.y = kr_nc;

  grid_lr_edge.x = (in_nr - kr_nr + int(Dev_props::max_threads_per_block_x())) /
                                    int(Dev_props::max_threads_per_block_x());
  grid_lr_edge.y = 1;

  grid_tb_edge.x = 1;
  grid_tb_edge.y = (in_nc - kr_nc + int(Dev_props::max_threads_per_block_y())) /
                                    int(Dev_props::max_threads_per_block_y());

  threads_lr_edge.x = int(Dev_props::max_threads_per_block_x());
  threads_lr_edge.y = kr_nc;

  threads_tb_edge.x = kr_nr;
  threads_tb_edge.y = int(Dev_props::max_threads_per_block_y());

  k_corr2dnd<<<grid, threads, shared_memory_launch_size>>>(
               in, kr, out, int(in_nr), int(in_nc), int(kr_nr), int(kr_nc),
               int(ou_nr), int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               center, same_supprt_nrows_even_ncols_odd);

  k_corr2dnd<<<grid_corner, threads_corner, shared_memory_launch_size>>>(
               in, kr, out, int(in_nr), int(in_nc), int(kr_nr), int(kr_nc),
               int(ou_nr), int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               corners, same_supprt_nrows_even_ncols_odd);

  k_corr2dnd<<<grid_lr_edge, threads_lr_edge, shared_memory_launch_size>>>(
               in, kr, out, int(in_nr), int(in_nc), int(kr_nr), int(kr_nc),
               int(ou_nr), int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               left_right_edge, same_supprt_nrows_even_ncols_odd);

  k_corr2dnd<<<grid_tb_edge, threads_tb_edge, shared_memory_launch_size>>>(
               in, kr, out, int(in_nr), int(in_nc), int(kr_nr), int(kr_nc),
               int(ou_nr), int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               top_bottom_edge, same_supprt_nrows_even_ncols_odd);

}

void
corr_2d_no_decimation_same_nrow_even_ncol_odd(
  std::complex<float> const*        in,
  std::complex<float> const*        kr,
  std::complex<float>*              out,
  size_t              in_nr,
  size_t              in_nc,
  size_t              kr_nr,
  size_t              kr_nc,
  size_t              ou_nr,
  size_t              ou_nc,
  size_t              rshift,
  size_t              cshift,
  int                 bias)
{
  dim3 grid, threads, grid_corner, threads_corner, grid_lr_edge,
       threads_lr_edge, grid_tb_edge, threads_tb_edge;

  int dtr = -rshift - 1;
  int dtc = -cshift;

  size_t const shared_memory_launch_size = 16000;

  threads.x = min(int(Dev_props::max_threads_per_block_x()),
                  int(sqrt(float(Dev_props::shared_memory_size() - 72) /
                                (8.0 * 2.5))));
  threads.y = threads.x / 2;

  grid.x = (in_nr - kr_nr + threads.x) / threads.x;
  grid.y = (in_nc - kr_nc + threads.y) / threads.y;

  grid_corner.x = 1;
  grid_corner.y = 1;
  threads_corner.x = kr_nr;
  threads_corner.y = kr_nc;

  grid_lr_edge.x = grid.x;
  grid_lr_edge.y = 1;

  grid_tb_edge.x = 1;
  grid_tb_edge.y = grid.y;

  threads_lr_edge.x = threads.x;
  threads_lr_edge.y = kr_nc;

  threads_tb_edge.x = kr_nr;
  threads_tb_edge.y = threads.y;

  k_corr2dnd<<<grid, threads, shared_memory_launch_size>>>(
               reinterpret_cast<hipComplex const*>(in),
               reinterpret_cast<hipComplex const*>(kr),
               reinterpret_cast<hipComplex*>(out), int(in_nr),
               int(in_nc), int(kr_nr), int(kr_nc), int(ou_nr),
               int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               center, same_supprt_nrows_even_ncols_odd);

  k_corr2dnd<<<grid_corner, threads_corner, shared_memory_launch_size>>>(
               reinterpret_cast<hipComplex const*>(in),
               reinterpret_cast<hipComplex const*>(kr),
               reinterpret_cast<hipComplex*>(out), int(in_nr),
               int(in_nc), int(kr_nr), int(kr_nc), int(ou_nr),
               int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               corners, same_supprt_nrows_even_ncols_odd);

  k_corr2dnd<<<grid_lr_edge, threads_lr_edge, shared_memory_launch_size>>>(
               reinterpret_cast<hipComplex const*>(in),
               reinterpret_cast<hipComplex const*>(kr),
               reinterpret_cast<hipComplex*>(out), int(in_nr),
               int(in_nc), int(kr_nr), int(kr_nc), int(ou_nr),
               int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               left_right_edge, same_supprt_nrows_even_ncols_odd);

  k_corr2dnd<<<grid_tb_edge, threads_tb_edge, shared_memory_launch_size>>>(
               reinterpret_cast<hipComplex const*>(in),
               reinterpret_cast<hipComplex const*>(kr),
               reinterpret_cast<hipComplex*>(out), int(in_nr),
               int(in_nc), int(kr_nr), int(kr_nc), int(ou_nr),
               int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               top_bottom_edge, same_supprt_nrows_even_ncols_odd);

}
void
corr_2d_no_decimation_same_nrow_odd_ncol_even(
  float const*        in,
  float const*        kr,
  float*              out,
  size_t              in_nr,
  size_t              in_nc,
  size_t              kr_nr,
  size_t              kr_nc,
  size_t              ou_nr,
  size_t              ou_nc,
  size_t              rshift,
  size_t              cshift,
  int                 bias)
{
  dim3 grid, threads, grid_corner, threads_corner, grid_lr_edge,
       threads_lr_edge, grid_tb_edge, threads_tb_edge;

  int dtr = -rshift;
  int dtc = -cshift - 1;

  size_t const shared_memory_launch_size = 16000;

  distribute_matrix(in_nc - kr_nc + 1, in_nr - kr_nr + 1, grid, threads);
  grid_corner.x = 1;
  grid_corner.y = 1;
  threads_corner.x = kr_nr;
  threads_corner.y = kr_nc;

  grid_lr_edge.x = (in_nr - kr_nr + int(Dev_props::max_threads_per_block_x())) /
                                    int(Dev_props::max_threads_per_block_x());
  grid_lr_edge.y = 1;

  grid_tb_edge.x = 1;
  grid_tb_edge.y = (in_nc - kr_nc + int(Dev_props::max_threads_per_block_y())) /
                                    int(Dev_props::max_threads_per_block_y());

  threads_lr_edge.x = int(Dev_props::max_threads_per_block_x());
  threads_lr_edge.y = kr_nc;

  threads_tb_edge.x = kr_nr;
  threads_tb_edge.y = int(Dev_props::max_threads_per_block_y());

  k_corr2dnd<<<grid, threads, shared_memory_launch_size>>>(
               in, kr, out, int(in_nr), int(in_nc), int(kr_nr), int(kr_nc),
               int(ou_nr), int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               center, same_supprt_nrows_odd_ncols_even);

  k_corr2dnd<<<grid_corner, threads_corner, shared_memory_launch_size>>>(
               in, kr, out, int(in_nr), int(in_nc), int(kr_nr), int(kr_nc),
               int(ou_nr), int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               corners, same_supprt_nrows_odd_ncols_even);

  k_corr2dnd<<<grid_lr_edge, threads_lr_edge, shared_memory_launch_size>>>(
               in, kr, out, int(in_nr), int(in_nc), int(kr_nr), int(kr_nc),
               int(ou_nr), int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               left_right_edge, same_supprt_nrows_odd_ncols_even);

  k_corr2dnd<<<grid_tb_edge, threads_tb_edge, shared_memory_launch_size>>>(
               in, kr, out, int(in_nr), int(in_nc), int(kr_nr), int(kr_nc),
               int(ou_nr), int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               top_bottom_edge, same_supprt_nrows_odd_ncols_even);

}

void
corr_2d_no_decimation_same_nrow_odd_ncol_even(
  std::complex<float> const*        in,
  std::complex<float> const*        kr,
  std::complex<float>*              out,
  size_t              in_nr,
  size_t              in_nc,
  size_t              kr_nr,
  size_t              kr_nc,
  size_t              ou_nr,
  size_t              ou_nc,
  size_t              rshift,
  size_t              cshift,
  int                 bias)
{
  dim3 grid, threads, grid_corner, threads_corner, grid_lr_edge,
       threads_lr_edge, grid_tb_edge, threads_tb_edge;

  int dtr = -rshift;
  int dtc = -cshift - 1;

  size_t const shared_memory_launch_size = 16000;

  threads.x = min(int(Dev_props::max_threads_per_block_x()),
                  int(sqrt(float(Dev_props::shared_memory_size() - 72) /
                                (8.0 * 2.5))));
  threads.y = threads.x / 2;

  grid.x = (in_nr - kr_nr + threads.x) / threads.x;
  grid.y = (in_nc - kr_nc + threads.y) / threads.y;


  grid_corner.x = 1;
  grid_corner.y = 1;
  threads_corner.x = kr_nr;
  threads_corner.y = kr_nc;

  grid_lr_edge.x = grid.x;
  grid_lr_edge.y = 1;

  grid_tb_edge.x = 1;
  grid_tb_edge.y = grid.y;

  threads_lr_edge.x = threads.x;
  threads_lr_edge.y = kr_nc;

  threads_tb_edge.x = kr_nr;
  threads_tb_edge.y = threads.y;

  k_corr2dnd<<<grid, threads, shared_memory_launch_size>>>(
               reinterpret_cast<hipComplex const*>(in),
               reinterpret_cast<hipComplex const*>(kr),
               reinterpret_cast<hipComplex*>(out), int(in_nr),
               int(in_nc), int(kr_nr), int(kr_nc), int(ou_nr),
               int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               center, same_supprt_nrows_odd_ncols_even);

  k_corr2dnd<<<grid_corner, threads_corner, shared_memory_launch_size>>>(
               reinterpret_cast<hipComplex const*>(in),
               reinterpret_cast<hipComplex const*>(kr),
               reinterpret_cast<hipComplex*>(out), int(in_nr),
               int(in_nc), int(kr_nr), int(kr_nc), int(ou_nr),
               int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               corners, same_supprt_nrows_odd_ncols_even);

  k_corr2dnd<<<grid_lr_edge, threads_lr_edge, shared_memory_launch_size>>>(
               reinterpret_cast<hipComplex const*>(in),
               reinterpret_cast<hipComplex const*>(kr),
               reinterpret_cast<hipComplex*>(out), int(in_nr),
               int(in_nc), int(kr_nr), int(kr_nc), int(ou_nr),
               int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               left_right_edge, same_supprt_nrows_odd_ncols_even);

  k_corr2dnd<<<grid_tb_edge, threads_tb_edge, shared_memory_launch_size>>>(
               reinterpret_cast<hipComplex const*>(in),
               reinterpret_cast<hipComplex const*>(kr),
               reinterpret_cast<hipComplex*>(out), int(in_nr),
               int(in_nc), int(kr_nr), int(kr_nc), int(ou_nr),
               int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               top_bottom_edge, same_supprt_nrows_odd_ncols_even);
}

void
corr_2d_no_decimation_same_nrow_odd_ncol_odd(
  float const*        in,
  float const*        kr,
  float*              out,
  size_t              in_nr,
  size_t              in_nc,
  size_t              kr_nr,
  size_t              kr_nc,
  size_t              ou_nr,
  size_t              ou_nc,
  size_t              rshift,
  size_t              cshift,
  int                 bias)
{
  dim3 grid, threads, grid_corner, threads_corner, grid_lr_edge,
       threads_lr_edge, grid_tb_edge, threads_tb_edge;

  int dtr = -rshift;
  int dtc = -cshift;

  size_t const shared_memory_launch_size = 16000;

  distribute_matrix(in_nc - kr_nc + 1, in_nr - kr_nr + 1, grid, threads);
  
  grid_corner.x = 1;
  grid_corner.y = 1;
  threads_corner.x = kr_nr;
  threads_corner.y = kr_nc;

  grid_lr_edge.x = (in_nr - kr_nr + int(Dev_props::max_threads_per_block_x())) /
                                    int(Dev_props::max_threads_per_block_x());
  grid_lr_edge.y = 1;

  grid_tb_edge.x = 1;
  grid_tb_edge.y = (in_nc - kr_nc + int(Dev_props::max_threads_per_block_y())) /
                                    int(Dev_props::max_threads_per_block_y());

  threads_lr_edge.x = int(Dev_props::max_threads_per_block_x());
  threads_lr_edge.y = kr_nc;

  threads_tb_edge.x = kr_nr;
  threads_tb_edge.y = int(Dev_props::max_threads_per_block_y());

  k_corr2dnd<<<grid, threads, shared_memory_launch_size>>>(
               in, kr, out, int(in_nr), int(in_nc), int(kr_nr), int(kr_nc),
               int(ou_nr), int(ou_nc), int(rshift), int(cshift), int(dtr),
               int(dtc), bias, center, same_supprt_nrows_odd_ncols_odd);

  k_corr2dnd<<<grid_corner, threads_corner, shared_memory_launch_size>>>(
               in, kr, out, int(in_nr), int(in_nc), int(kr_nr), int(kr_nc),
               int(ou_nr), int(ou_nc), int(rshift), int(cshift), int(dtr),
               int(dtc), bias, corners, same_supprt_nrows_odd_ncols_odd);

  k_corr2dnd<<<grid_lr_edge, threads_lr_edge, shared_memory_launch_size>>>(
               in, kr, out, int(in_nr), int(in_nc), int(kr_nr), int(kr_nc),
               int(ou_nr), int(ou_nc), int(rshift), int(cshift), int(dtr),
               int(dtc), bias, left_right_edge, same_supprt_nrows_odd_ncols_odd);

  k_corr2dnd<<<grid_tb_edge, threads_tb_edge, shared_memory_launch_size>>>(
               in, kr, out, int(in_nr), int(in_nc), int(kr_nr), int(kr_nc),
               int(ou_nr), int(ou_nc), int(rshift), int(cshift), int(dtr),
               int(dtc), bias, top_bottom_edge, same_supprt_nrows_odd_ncols_odd);

}

void
corr_2d_no_decimation_same_nrow_odd_ncol_odd(
  std::complex<float> const*        in,
  std::complex<float> const*        kr,
  std::complex<float>*              out,
  size_t                            in_nr,
  size_t                            in_nc,
  size_t                            kr_nr,
  size_t                            kr_nc,
  size_t                            ou_nr,
  size_t                            ou_nc,
  size_t                            rshift,
  size_t                            cshift,
  int                 bias)
{
  dim3 grid, threads, grid_corner, threads_corner, grid_lr_edge,
       threads_lr_edge, grid_tb_edge, threads_tb_edge;

  int dtr = -rshift;
  int dtc = -cshift;

  size_t const shared_memory_launch_size = 16000;

  threads.x = min(int(Dev_props::max_threads_per_block_x()),
                  int(sqrt(float(Dev_props::shared_memory_size() - 72) /
                                (8.0 * 2.5))));
  threads.y = threads.x / 2;

  grid.x = (in_nr - kr_nr + threads.x) / threads.x;
  grid.y = (in_nc - kr_nc + threads.y) / threads.y;


  grid_corner.x = 1;
  grid_corner.y = 1;
  threads_corner.x = kr_nr;
  threads_corner.y = kr_nc;

  grid_lr_edge.x = grid.x;
  grid_lr_edge.y = 1;

  grid_tb_edge.x = 1;
  grid_tb_edge.y = grid.y;

  threads_lr_edge.x = threads.x;
  threads_lr_edge.y = kr_nc;

  threads_tb_edge.x = kr_nr;
  threads_tb_edge.y = threads.y;

  k_corr2dnd<<<grid, threads, shared_memory_launch_size>>>(
               reinterpret_cast<hipComplex const*>(in),
               reinterpret_cast<hipComplex const*>(kr),
               reinterpret_cast<hipComplex*>(out), int(in_nr),
               int(in_nc), int(kr_nr), int(kr_nc), int(ou_nr),
               int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               center, same_supprt_nrows_odd_ncols_odd);

  k_corr2dnd<<<grid_corner, threads_corner, shared_memory_launch_size>>>(
               reinterpret_cast<hipComplex const*>(in),
               reinterpret_cast<hipComplex const*>(kr),
               reinterpret_cast<hipComplex*>(out), int(in_nr),
               int(in_nc), int(kr_nr), int(kr_nc), int(ou_nr),
               int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               corners, same_supprt_nrows_odd_ncols_odd);

  k_corr2dnd<<<grid_lr_edge, threads_lr_edge, shared_memory_launch_size>>>(
               reinterpret_cast<hipComplex const*>(in),
               reinterpret_cast<hipComplex const*>(kr),
               reinterpret_cast<hipComplex*>(out), int(in_nr),
               int(in_nc), int(kr_nr), int(kr_nc), int(ou_nr),
               int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               left_right_edge, same_supprt_nrows_odd_ncols_odd);

  k_corr2dnd<<<grid_tb_edge, threads_tb_edge, shared_memory_launch_size>>>(
               reinterpret_cast<hipComplex const*>(in),
               reinterpret_cast<hipComplex const*>(kr),
               reinterpret_cast<hipComplex*>(out), int(in_nr),
               int(in_nc), int(kr_nr), int(kr_nc), int(ou_nr),
               int(ou_nc), int(rshift), int(cshift), dtr, dtc, bias,
               top_bottom_edge, same_supprt_nrows_odd_ncols_odd);

}

} // namespace vsip::impl::cuda
} // namespace vsip::impl
} // namespace vsip
