#include "hip/hip_runtime.h"
/* Copyright (c) 2010 by CodeSourcery.  All rights reserved. */

/// Description
///   CUDA kernel for atan.

#include <hip/hip_complex.h>
#include "util.hpp"
#include <vsip/support.hpp>
#include <complex>

namespace dev
{
// 1-D unit stride
__global__ void 
atan(float const* in, float* out, size_t length)
{
  int const tx = threadIdx.x;
  int const bx = blockIdx.x;

  int const idx = __mul24(blockDim.x, bx) + tx;
  if (idx < length)
    out[idx] = atanf(in[idx]);
}

// 1-D general stride
__global__ void 
atan(float const* in, ptrdiff_t in_stride, float* out, ptrdiff_t out_stride, size_t length)
{
  int const tx = threadIdx.x;
  int const bx = blockIdx.x;

  int const tid = __mul24(blockDim.x, bx) + tx;
  int const in_idx = __mul24(tid, in_stride);
  int const out_idx = __mul24(tid, out_stride);
  if (tid < length)
    out[out_idx] = atanf(in[in_idx]);
}

// 2-D general stride
__global__ void 
atan(
  float const* in,
  ptrdiff_t    row_in_stride,
  ptrdiff_t    col_in_stride,
  float*       out,
  ptrdiff_t    row_out_stride,
  ptrdiff_t    col_out_stride,
  size_t       num_rows,
  size_t       num_cols)
{
  int const tx = threadIdx.x;
  int const ty = threadIdx.y;
  int const bx = blockIdx.x;
  int const by = blockIdx.y;

  int const tidx = __mul24(blockDim.x, bx) + tx;
  int const tidy = __mul24(blockDim.y, by) + ty;
  int const in_idx = __mul24(tidy, row_in_stride) + __mul24(tidx, col_in_stride);
  int const out_idx = __mul24(tidy, row_out_stride) + __mul24(tidx, col_out_stride);
  if (tidy < num_rows && tidx < num_cols)
    out[out_idx] = atanf(in[in_idx]);
}
}// namespace dev

namespace vsip
{
namespace impl
{
namespace cuda
{
void
atan(
  float const*     in,
  float*           out,
  length_type      length)
{
  dim3 grid, threads;
  distribute_vector(length, grid, threads);

  dev::atan<<<grid, threads>>>(in, out, length);
}

void
atan(
  float const*     in,
  stride_type      in_stride,
  float*           out,
  stride_type      out_stride,
  length_type      length)
{
  dim3 grid, threads;
  distribute_vector(length, grid, threads);

  dev::atan<<<grid, threads>>>(in, in_stride, out, out_stride, length);
}

void
atan(
  float const*     in,
  stride_type      row_in_stride,
  stride_type      col_in_stride,
  float*           out,
  stride_type      row_out_stride,
  stride_type      col_out_stride,
  length_type      num_rows,
  length_type      num_cols)
{
  dim3 grid, threads;
  distribute_matrix(num_rows, num_cols, grid, threads);

  dev::atan<<<grid, threads>>>(in, row_in_stride, col_in_stride,
                               out, row_out_stride, col_out_stride,
                               num_rows, num_cols);
}
} // namespace vsip::impl::cuda
} // namespace vsip::impl
} // namespace vsip
